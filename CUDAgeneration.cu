#include "hip/hip_runtime.h"
#ifndef CUDA_EVO_ALL

#define CUDA_EVO_ALL

#include "life.hpp"

/*
======================================================
|| void cu_evolve()
======================================================
|| same as normal evolve exepct run on device (GPU)
|| all extranl function used were put into this function
|| to simplify GPU code
======================================================
|| state1 -> state that holds the current world state
|| state2 -> state to use a temproy storgae for future state
|| size -> the size to use for hashing
======================================================
|| returns and updated state1
======================================================
*/
__global__ void cu_evolve(unsigned* state1, unsigned* state2, const unsigned size) {
    //set up where to start and go for this specifc run based on where it is in the GPU thread block grid
    int idx = (blockDim.x*blockDim.y)*threadIdx.z+(threadIdx.y*blockDim.x)+(threadIdx.x); //thread id in block
    int bdx = (gridDim.x*gridDim.y)*blockIdx.z +(blockIdx.y*gridDim.x)+(blockIdx.x); //block id in grid
    int bsize = blockDim.x*blockDim.y*blockDim.z; //block size
    int gsize = gridDim.x*gridDim.y*gridDim.z; //grid size
    //frist one needs to start at 1 since the world as a one space border
    if (bdx == 0) {
        bdx = 1;
    }
    // main update loop
    for (int y = bdx; y < size; y+= gsize){
        for (int x = idx; x < size; x += bsize){
            u_int8_t window[WINDOWSIZE];

            //expanded resetWindow()
            int left;
            int right;
            int up;
            int down;
            if (x == 1) {
                left = size - 1;
                right = x + 1;
            }
            else if (x == size - 1) {
                left = x - 1;
                right = 1;
            }
            else {
                left = x - 1;
                right = x + 1;
            }

            if (y == 1) {
                up = size - 1;
                down = y + 1;
            }
            else if (y == size - 1) {
                up = y - 1;
                down = 1;
            }
            else {
                up = y - 1;
                down = y + 1;
            }
            window[0] = state1[up*size+left];
            window[1] = state1[up*size+x];
            window[2] = state1[up*size+right];
            window[3] = state1[y*size+left];
            window[4] = state1[y*size+x];
            window[5] = state1[y*size+right];
            window[6] = state1[down*size+left];
            window[7] = state1[down*size+x];
            window[8] = state1[down*size+right];
            unsigned weight = 0;
            //end of resetWindow()

            //expanded calculateWeight()
            for (int i = 0; i < WINDOWSIZE; i++){
                weight += window[i];
            }
            int updated = 0;
            weight -= window[4];
            //end of calculateWeight()

            //expanded testWeight() with generalised ruleset
            //removed to simplify arguments that need to be moved ot GPU memory
            if (state1[y*size+x] == 1){
                for (unsigned i : {2,3}){
                    if (weight == i){
                        state2[y*size+x] = 1;
                        updated = 1;
                    }
                }
            }
            else if (state1[y*size+x] == 0){
                for (unsigned i : {3}){
                    if (weight == i){
                        state2[y*size+x] = 1;
                        updated = 1;
                    }
                }
            }
            if (updated == 0) { //add this terms since using C arrays instead of Vectors
                state2[y*size+x] = 0;
            }

            //end of testWeight()
        }
    }
    __syncthreads(); //not really necassary
}

void cuda_evo_all(VectorState& state1, VectorState& state2, unsigned size, unsigned generations) {
    //timer setup
    std::chrono::time_point<std::chrono::system_clock> start, end;
    std::time_t end_time = std::chrono::system_clock::to_time_t(end);

    end = std::chrono::system_clock::now();
    std::cout << "started copying to GPU memory " << std::ctime(&end_time);

    //convert Vector states into 1D C arrays to move to Device
    unsigned* arr_state1 = (unsigned*) malloc(sizeof(unsigned) * size * size);
    unsigned* arr_state2 = (unsigned*) malloc(sizeof(unsigned) * size * size);

    for (unsigned i = 0; i < size; i++) {
        for (unsigned j = 0; j < size; j++) {
            arr_state1[i*size+j] = state1[i][j];
            arr_state2[i*size+j] = state2[i][j];
        }
    }

    //move arrays to device memory from cpu memory
    unsigned* custate1;
    unsigned* custate2;

    hipMalloc((void**)&custate1, sizeof(unsigned) * size * size);
    hipMalloc((void**)&custate2, sizeof(unsigned) * size * size);
    hipMemcpy(custate1, arr_state1, sizeof(unsigned) * size * size, hipMemcpyHostToDevice);
    hipMemcpy(custate2, arr_state2, sizeof(unsigned) * size * size, hipMemcpyHostToDevice);

    //end og mem move timing
    end = std::chrono::system_clock::now();
    end_time = std::chrono::system_clock::to_time_t(end);
    std::cout << "finished copying to GPU memory " << std::ctime(&end_time);
    std::cout<<"======================================================\n";

    start = std::chrono::system_clock::now();

    //normal gen ervolve loop
    for (int i = 0; i < generations; i++){
        //std::cout << "hello " << i << "\n";
        if (i%(generations/FRAMES) == 0 && i != 0){
            end = std::chrono::system_clock::now();
            std::chrono::duration<double> elapsed_seconds = end - start; //output time after computing 10% of calcualtions
            std::cout << "| (" << i << "/" << generations << "):   " << "elapsed time: " << elapsed_seconds.count() << "s\n";
            //exportState(state1); //export to file
        }
        cu_evolve<<<32, 1024>>>(custate1, custate2, size); //run device with 1024x32 grid
        hipDeviceSynchronize(); //synchronise since previous gens depend on furture gens
        unsigned* temp;
        temp = custate1; //swaps
        custate1 = custate2;
        custate2 = temp;
    }

    std::cout<<"======================================================\n";
    end = std::chrono::system_clock::now();
    std::cout << "started copying from gpu memory to cpu memory " << std::ctime(&end_time);

    //vopy array back of device to cpu so it can be output
    hipMemcpy(arr_state1, custate1, sizeof(unsigned) * size * size, hipMemcpyDeviceToHost);

    //convert back to vector
    for (unsigned i = 0; i < size; i++) {
        for (unsigned j = 0; j < size; j++) {
            state1[i][j] = arr_state1[i*size+j];
        }
    }

    //free mem
    free(arr_state1);
    free(arr_state2);
    hipFree(custate1);
    hipFree(custate2);

    end = std::chrono::system_clock::now();
    end_time = std::chrono::system_clock::to_time_t(end);
    std::cout << "finished copying from gpu memory to cpu memory " << std::ctime(&end_time);
}

#endif